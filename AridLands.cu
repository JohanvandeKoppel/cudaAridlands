#include "hip/hip_runtime.h"
/*
ROHIT GUPTA & JOHAN VAN DE KOPPEL
Arid Pattern formation
June 2010
*/

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// include CUDA
#include <hip/hip_runtime.h>

// include parameter kernel
#include "AridLands.h"

////////////////////////////////////////////////////////////////////////////////
// Allocates a matrix with random float entries
////////////////////////////////////////////////////////////////////////////////

void randomInit(float* data, int x_siz, int y_siz, int type)
{
	int i,j;
	for(i=0;i<y_siz;i++)
	{
		for(j=0;j<x_siz;j++)
		{
			//for every element find the correct initial
			//value using the conditions below
			if(i==0||i==y_siz-1||j==0||j==x_siz-1)
			    data[i*x_siz+j]=0.0f; // This value for the boundaries
			else
			{
				if(type==Plants)
					
				{
                    // A randomized initiation here
					if((rand() / (float)RAND_MAX)<0.05f)
							data[i*x_siz+j] = 100.0f;
						else
							data[i*x_siz+j] = 0.0f;
				}
				else if(type==Surface_Water)
					data[i*x_siz+j]=(float)(R/(alpha*W0));
				else if(type==Soil_Water)
					data[i*x_siz+j]=(float)(R/rw/4);
			}
		}
	}			

} // End randomInit

////////////////////////////////////////////////////////////////////////////////
// Laplacation operator definition, to calculate diffusive fluxes
////////////////////////////////////////////////////////////////////////////////

__device__ float
LaplacianXY(float* pop, int row, int column)
{
	float retval;
	int current, left, right, top, bottom;	
	float dx = dX;
	float dy = dY;
	
	current=row * WidthGrid + column;	
	left=row * WidthGrid + column-1;
	right=row * WidthGrid + column+1;
	top=(row-1) * WidthGrid + column;
	bottom=(row+1) * WidthGrid + column;

	retval = ( (( pop[current] - pop[left] )/dx ) 
		      -(( pop[right] - pop[current] )/dx )) / dx + 
		     ( (( pop[current] - pop[top] )/dy  ) 
			  -(( pop[bottom] - pop[current] )/dy ) ) / dy;

	return retval;
}

////////////////////////////////////////////////////////////////////////////////
// Gradient operator definition, to calculate advective fluxes
////////////////////////////////////////////////////////////////////////////////


__device__ float
GradientY(float* pop, int row, int column)
{
	float retval;
	int current, top;	
	float dy =dY;
	
	current=row * WidthGrid + column;	
	top=(row-1) * WidthGrid + column;
	
	retval =  (( pop[current] - pop[top] )/dy ); 

	return retval;
}

////////////////////////////////////////////////////////////////////////////////
// Simulation kernel
////////////////////////////////////////////////////////////////////////////////

__global__ void 
Aridlandskernel (float* popP, float* popW, float* popO)
{

	//run for U X V times. For every U times completed store in the array storeA and storeM

	int current;

	float d2popPdxy2, d2popWdxy2, d2popOdxy2;
	float drP,drW, drO;
	
	int row		=	blockIdx.y*Block_Size_Y+threadIdx.y;
	int column	=	blockIdx.x*Block_Size_X+threadIdx.x;
	
	current=row * WidthGrid + column;
	
	if(row > 0 && row < HeightGrid-1 && column > 0 && column < WidthGrid-1)
	 {		
		//Now calculating terms for the O Matrix	
		d2popOdxy2 = -DifO * LaplacianXY(popO, row, column) - AdvO * GradientY(popO, row, column);
		drO = (R-alpha*(popP[current]+k2*W0)/(popP[current]+k2)*popO[current]);
		 
		//Now calculating terms for the W Matrix
		d2popWdxy2 = -DifW * LaplacianXY(popW, row, column);
		drW = (alpha*(popP[current]+k2*W0)/(popP[current]+k2)*popO[current] 
	          - gmax*popW[current]/(popW[current]+k1)*popP[current]-rw*popW[current]);

		//Now calculating terms for the P Matrix
		d2popPdxy2 = -DifP * LaplacianXY(popP, row, column);
		drP = (c*gmax*popW[current]/(popW[current]+k1)*popP[current] - d*popP[current]);

		__syncthreads();

		popO[current]=popO[current]+(drO+d2popOdxy2)*dT;		
		popW[current]=popW[current]+(drW+d2popWdxy2)*dT;	
		popP[current]=popP[current]+(drP+d2popPdxy2)*dT;
	
	 }

	__syncthreads();

	// HANDLE Boundaries
	if(row==0)
			{
				popW[row * WidthGrid + column]=popW[(HeightGrid-2) * WidthGrid+column];
				popO[row * WidthGrid + column]=popO[(HeightGrid-2) * WidthGrid+column];
				popP[row * WidthGrid + column]=popP[(HeightGrid-2) * WidthGrid+column];
			}
	else if(row==HeightGrid-1)			
			{
				popW[row * WidthGrid + column]=popW[1*WidthGrid+column];
				popO[row * WidthGrid + column]=popO[1*WidthGrid+column];
				popP[row * WidthGrid + column]=popP[1*WidthGrid+column];
			}	
	else if(column==0)			
			{
				popW[row * WidthGrid + column]=popW[row * WidthGrid + WidthGrid-2];
				popO[row * WidthGrid + column]=popO[row * WidthGrid + WidthGrid-2];
				popP[row * WidthGrid + column]=popP[row * WidthGrid + WidthGrid-2];
			}	
	else if(column==WidthGrid-1)			
			{
				popW[row * WidthGrid + column]=popW[row * WidthGrid + 1];
				popO[row * WidthGrid + column]=popO[row * WidthGrid + 1];
				popP[row * WidthGrid + column]=popP[row * WidthGrid + 1];
			}	
	
} // End Aridlandskernel

////////////////////////////////////////////////////////////////////////////////
// Main program
////////////////////////////////////////////////////////////////////////////////
int
main(int argc, char** argv)
{
	int store_count;       // The number of times a frame was stored
	double time_elapsed;   // The amount of time that has passed

	int NumStored;
	int store_i,i;
	unsigned int size_grid = WidthGrid * HeightGrid;
	unsigned int mem_size_grid = sizeof(float) * size_grid;
	unsigned int size_storegrid = WidthGrid * HeightGrid * MAX_STORE;
	unsigned int mem_size_storegrid = sizeof(float) * size_storegrid;

    double RecordTimes[MAX_STORE];
    
	float* h_store_popP;
	float* h_store_popO;
	float* h_store_popW;

	float* h_popP;
	float* h_popO;
	float* h_popW;

	float* d_popP;
	float* d_popO;
	float* d_popW;
    
	FILE *fp;
    
	int height_matrix=HeightGrid;
	int width_matrix=WidthGrid;
    
    /*--------------------INITIALIZATIONS ON HOST-------------------*/
	time_elapsed=Time;
	store_count=0;

	// set seed for rand()
	srand((unsigned)time( NULL ));

	//allocate host memory for matrices popP, popO, and popW
	h_popP = (float*) malloc(mem_size_grid);
	h_popO = (float*) malloc(mem_size_grid);
	h_popW = (float*) malloc(mem_size_grid);	

	//allocate host memory for matrices store_popP, store_popO, and store_popW
	h_store_popP=(float*) malloc(mem_size_storegrid);
	h_store_popO=(float*) malloc(mem_size_storegrid);
	h_store_popW=(float*) malloc(mem_size_storegrid);

    /*---------------------INITIALIZING THE ARRAYS----------------------------*/
	randomInit(h_popP, WidthGrid, HeightGrid, Plants);
	randomInit(h_popO, WidthGrid, HeightGrid, Surface_Water);
	randomInit(h_popW, WidthGrid, HeightGrid, Soil_Water);

    /*---------------------INITIALIZING ON GPU--------------------------------*/
	
        // allocate device memory    
	hipMalloc((void**) &d_popP, mem_size_grid);
	hipMalloc((void**) &d_popO, mem_size_grid);
	hipMalloc((void**) &d_popW, mem_size_grid);

        //copy host memory to device
	hipMemcpy(d_popP, h_popP, mem_size_grid, hipMemcpyHostToDevice);
	hipMemcpy(d_popO, h_popO, mem_size_grid, hipMemcpyHostToDevice);
	hipMemcpy(d_popW, h_popW, mem_size_grid, hipMemcpyHostToDevice);   
 
    /*---------------------SETUP EXECUTION PARAMETERS-------------------------*/	
	dim3 threads;      // Setting up the GPU setting, thread block size
	dim3 grid;         // Setting up the GPU setting, grid structure

	threads.x= Block_Size_X;
	threads.y= Block_Size_Y;
	grid.x=DIVIDE_INTO(WidthGrid,threads.x);
	grid.y=DIVIDE_INTO(HeightGrid,threads.y);

    //using namespace std;
    clock_t begin = clock();

    // Calculate the times at which the simulation is stored
    for(i=0;i<=NumFrames;i++) 
		{ RecordTimes[i]=(double)i/(double)NumFrames*(double)EndTime; }  

    /*----- Printing info to the screen --------------------------------*/
	system("clear");
        printf("\n");
	printf(" * * * * * * * * * * * * * * * * * * * * * * * * * * * * * \n");
	printf(" * Arid Land Patters                                     * \n");		
	printf(" * CUDA implementation : Rohit Gupta, 2009               * \n");
	printf(" * Following a model by Rietkerk et al 2002              * \n");
	printf(" * * * * * * * * * * * * * * * * * * * * * * * * * * * * * \n\n");
	
	int device;
    hipGetDevice(&device);

    struct hipDeviceProp_t props;
    hipGetDeviceProperties(&props, device);
    
    printf(" Using device: %s \n\n", props.name);

	printf(" Current grid dimensions: %d x %d cells\n\n", WidthGrid, HeightGrid);
    
    /*----- The simulation loop ----------------------------------------*/
	while((float)time_elapsed<=(float)EndTime)
	   {
		// execute the kernel
  		Aridlandskernel<<< grid, threads >>>(d_popP, d_popW, d_popO);

		// Storing the data if a particular number of timesteps have passed	
		if((float)time_elapsed>=(float)RecordTimes[store_count])
		    {		
			hipMemcpy((void *)h_popP, (void *)d_popP, mem_size_grid,
					   hipMemcpyDeviceToHost);
			hipMemcpy((void *)h_popO, (void *)d_popO, mem_size_grid,
					   hipMemcpyDeviceToHost);	
			hipMemcpy((void *)h_popW, (void *)d_popW, mem_size_grid,
					   hipMemcpyDeviceToHost);
			
			//Store values at this frame.
			memcpy(h_store_popP+(store_count*size_grid),h_popP,mem_size_grid);
			memcpy(h_store_popO+(store_count*size_grid),h_popO,mem_size_grid);
			memcpy(h_store_popW+(store_count*size_grid),h_popW,mem_size_grid);

			fprintf(stderr, "\r Current timestep: %1.0f of %1.0d, Storepoint %1d of %1d",
                                        time_elapsed,EndTime, store_count, NumFrames);
	
                        #if SaveEachPlot == on

			//	fp=fopen("CurrentFrame.dat","wb");	

			//	// Storing parameters
			//	fwrite(&width_matrix,sizeof(int),1,fp);
			//	fwrite(&height_matrix,sizeof(int),1,fp);

		        //      fwrite(&h_popP,sizeof(float),size_grid,fp);
                	//	fwrite(&h_popO,sizeof(float),size_grid,fp);
                	//	fwrite(&h_popW,sizeof(float),size_grid,fp);
	
			//	fclose(fp);
                        #endif	

			store_count=store_count+1;		
				
		    } // if on writing one frame ends
	
		time_elapsed=time_elapsed+(double)dT;
	
	   } //while on time ends
	
	/*---------------------Report on time spending----------------------------*/
	// sdkStopTimer(&timer);
	clock_t end = clock();
	//printf("\n\n %1.5f to %1.5f \n", (float)begin, (float)end);
    double elapsed_secs = (double)(end - begin)/CLOCKS_PER_SEC;
	printf("\n\n Processing time: %4.1f (s) \n", elapsed_secs);
	//sdkDeleteTimer(&timer);

	/*---------------------Write to file now----------------------------------*/
	fp=fopen("AridLands.dat","wb");	
	NumStored=store_count;	

	// Storing parameters
	fwrite(&width_matrix,sizeof(int),1,fp);
	fwrite(&height_matrix,sizeof(int),1,fp);
	fwrite(&NumStored,sizeof(int),1,fp);

    for(i=0;i<=NumStored;i++)
	   { fwrite(&RecordTimes[i],sizeof(double),1,fp); }	
	
	for(store_i=0;store_i<store_count;store_i++)
	   {
		fwrite(&h_store_popP[store_i*size_grid],sizeof(float),size_grid,fp);
		fwrite(&h_store_popO[store_i*size_grid],sizeof(float),size_grid,fp);
		fwrite(&h_store_popW[store_i*size_grid],sizeof(float),size_grid,fp);

		printf("\r Saving simulation results: %2.0f%%", (float)store_i/(float)store_count*100.0);
	   }
	
	printf("\r Saving simulation results: 100%%\n\n");

	fclose(fp);

	/*---------------------Clean up memory------------------------------------*/
	free(h_popP);
	free(h_popO);
	free(h_popW);

	free(h_store_popP);
	free(h_store_popO);
	free(h_store_popW);	
    
	hipFree(d_popP);
	hipFree(d_popO);
	hipFree(d_popW);

	hipDeviceReset();
	hipDeviceReset();
	
	#if defined(__APPLE__) && defined(__MACH__)
    system("say All ready");
    #endif

}
